
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_gpu(void)
{
    if (blockIdx.x == 0 && threadIdx.x > 3)
        return;
    printf("Hello from GPU%d[%d]!\n", blockIdx.x + 1, threadIdx.x);
}

int main()
{
    printf("Hello from CPU!\n");
    hello_gpu<<<2, 6>>>();

    hipDeviceSynchronize();
    return 0;
}