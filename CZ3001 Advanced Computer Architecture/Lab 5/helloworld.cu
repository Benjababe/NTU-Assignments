
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_gpu(void)
{
    // ignoring block 0 threads 4 and 5
    if (blockIdx.x == 0 && threadIdx.x > 3)
        return;
    printf("Hello from GPU%d[%d]!\n", blockIdx.x + 1, threadIdx.x);
}

int main()
{
    printf("Hello from CPU!\n");
    hello_gpu<<<2, 6>>>();

    hipDeviceSynchronize();
    return 0;
}