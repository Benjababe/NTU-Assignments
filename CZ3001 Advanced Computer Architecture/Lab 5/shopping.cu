
#include <hip/hip_runtime.h>
#include <stdio.h>

const int M = 4,
          N = 7;

// d_sales will be treated as 1D array because I'm too lazy to figure out what is a pitch for cudaMemcpy2D
__global__ void calc_sales(float *d_sales_figure, float *d_sales, float *d_prices)
{
    __shared__ float tmp[N * M];
    int i = threadIdx.x;

    tmp[i] = d_sales[i] * d_prices[i / N];
    __syncthreads();

    if (i == 0)
    {
        for (int j = 0; j < (M * N); j++)
            *d_sales_figure += tmp[j];
    }
}

int main()
{
    float sales[M][N] = {
        {3, 2, 0, 3, 4, 10, 8},  // HD
        {5, 4, 3, 5, 5, 13, 11}, // EP
        {2, 5, 3, 4, 5, 21, 15}, // SP
        {0, 1, 1, 4, 3, 16, 8}   // TD
    };

    float sales_figure = 0;
    float prices[M] = {29.99, 14.99, 9.99, 24.99};
    float *d_sales, *d_prices, *d_sales_figure;

    hipMalloc((void **)&d_sales, sizeof(float) * M * N);
    hipMalloc((void **)&d_prices, sizeof(float) * M);
    hipMalloc((void **)&d_sales_figure, sizeof(float));

    hipMemcpy(d_sales, sales, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipMemcpy(d_prices, prices, sizeof(float) * M, hipMemcpyHostToDevice);
    hipMemcpy(d_sales_figure, &sales_figure, sizeof(float), hipMemcpyHostToDevice);

    calc_sales<<<1, M * N>>>(d_sales_figure, d_sales, d_prices);
    hipDeviceSynchronize();

    hipMemcpy(&sales_figure, d_sales_figure, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_sales);
    hipFree(d_prices);
    hipFree(d_sales_figure);

    printf("Total sales: $%.2f\n", sales_figure);
    return 0;
}