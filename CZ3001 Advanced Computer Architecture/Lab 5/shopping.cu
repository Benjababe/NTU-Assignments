
#include <hip/hip_runtime.h>
#include <stdio.h>

const int M = 4,
          N = 7;

// d_sales will be treated as 1D array because I'm too lazy to figure out what is a pitch for cudaMemcpy2D
__global__ void calc_sales(float *d_sales_figure, float *d_sales, float *d_prices)
{
    __shared__ float tmp[N * M];
    int i = threadIdx.x;

    tmp[i] = d_sales[i] * d_prices[i / N];
    __syncthreads();

    // only thread 0 sums up sales values
    if (i == 0)
    {
        for (int j = 0; j < (M * N); j++)
            *d_sales_figure += tmp[j];
    }
}

int main()
{
    // for expansion, new item prices can just be appended to prices
    // and sales figures appended to sales as an array of the days
    float sales_figure = 0;
    float prices[M] = {29.99, 14.99, 9.99, 24.99};
    float sales[M][N] = {
        {3, 2, 0, 3, 4, 10, 8},  // HD
        {5, 4, 3, 5, 5, 13, 11}, // EP
        {2, 5, 3, 4, 5, 21, 15}, // SP
        {0, 1, 1, 4, 3, 16, 8}   // TD
    };

    // address pointers for vram
    float *d_sales, *d_prices, *d_sales_figure;

    // allocate video memory for values to be passed in
    hipMalloc((void **)&d_sales, sizeof(float) * M * N);
    hipMalloc((void **)&d_prices, sizeof(float) * M);
    hipMalloc((void **)&d_sales_figure, sizeof(float));

    // pass in our declared variables into vram
    // the 2D sales array is treated as a 1D array here
    // and sales_figure is just singular float value, not an array
    hipMemcpy(d_sales, sales, sizeof(float) * M * N, hipMemcpyHostToDevice);
    hipMemcpy(d_prices, prices, sizeof(float) * M, hipMemcpyHostToDevice);
    hipMemcpy(d_sales_figure, &sales_figure, sizeof(float), hipMemcpyHostToDevice);

    // everything will be done in a single block
    calc_sales<<<1, M * N>>>(d_sales_figure, d_sales, d_prices);
    hipDeviceSynchronize();

    // copy back the sales_figure float value to main memory
    hipMemcpy(&sales_figure, d_sales_figure, sizeof(float), hipMemcpyDeviceToHost);

    // trashman
    hipFree(d_sales);
    hipFree(d_prices);
    hipFree(d_sales_figure);

    // output
    printf("Total sales: $%.2f\n", sales_figure);
    return 0;
}