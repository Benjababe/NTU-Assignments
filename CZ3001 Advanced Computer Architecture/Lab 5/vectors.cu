
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 4;

__global__ void vector_add(int *h_c, int *h_a, int *h_b)
{
    int i = threadIdx.x;
    h_c[i] = h_a[i] + h_b[i];
}

__global__ void vector_dot_product(int *d_c, int *d_a, int *d_b)
{
    __shared__ int tmp[N];
    int i = threadIdx.x;
    tmp[i] = d_a[i] * d_b[i];

    __syncthreads();

    if (i == 0)
    {
        for (int j = 0; j < N; j++)
            *d_c += tmp[j];
    }
}

int main()
{
    int a[N] = {22, 13, 16, 5},
        b[N] = {5, 22, 17, 37},
        cAdd[N],
        cDot = 0;

    int *d_a, *d_b, *d_c, *d_c_dot;

    hipMalloc((void **)&d_a, sizeof(int) * N);
    hipMalloc((void **)&d_b, sizeof(int) * N);
    hipMalloc((void **)&d_c, sizeof(int) * N);

    hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, cAdd, sizeof(int) * N, hipMemcpyHostToDevice);

    vector_add<<<1, N>>>(d_c, d_a, d_b);
    hipDeviceSynchronize();

    hipMemcpy(cAdd, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipFree(d_c);

    printf("[Addition]\tOutput of C: { %d %d %d %d }\n", cAdd[0], cAdd[1], cAdd[2], cAdd[3]);

    hipMalloc((void **)&d_c_dot, sizeof(int));
    hipMemcpy(d_c_dot, &cDot, sizeof(int), hipMemcpyHostToDevice);

    vector_dot_product<<<1, N>>>(d_c, d_a, d_b);
    hipDeviceSynchronize();

    hipMemcpy(&cDot, d_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_c);

    printf("[Dot Product]\tOutput of C: %d", cDot);

    return 0;
}