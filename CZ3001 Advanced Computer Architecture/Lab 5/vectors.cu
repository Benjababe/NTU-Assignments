
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 4;

__global__ void vector_add(int *h_c, int *h_a, int *h_b)
{
    int i = threadIdx.x;
    h_c[i] = h_a[i] + h_b[i];
}

__global__ void vector_dot_product(int *d_c, int *d_a, int *d_b)
{
    __shared__ int tmp[N];
    int i = threadIdx.x;
    tmp[i] = d_a[i] * d_b[i];

    __syncthreads();

    // only thread 0 sums up multiplication values
    if (i == 0)
    {
        for (int j = 0; j < N; j++)
            *d_c += tmp[j];
    }
}

int main()
{
    int a[N] = {22, 13, 16, 5},
        b[N] = {5, 22, 17, 37},
        cAdd[N],
        cDot = 0;

    // address pointers for vram
    int *d_a, *d_b, *d_c, *d_c_dot;

    // allocate video memory for values to be passed in
    hipMalloc((void **)&d_a, sizeof(int) * N);
    hipMalloc((void **)&d_b, sizeof(int) * N);
    hipMalloc((void **)&d_c, sizeof(int) * N);

    // pass in our declared variables into vram
    hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, cAdd, sizeof(int) * N, hipMemcpyHostToDevice);

    // vector addition is done in 1 block
    vector_add<<<1, N>>>(d_c, d_a, d_b);
    hipDeviceSynchronize();

    // copy back the added matrix into main memory
    hipMemcpy(cAdd, d_c, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipFree(d_c);

    printf("[Addition]\tOutput of C: { %d %d %d %d }\n", cAdd[0], cAdd[1], cAdd[2], cAdd[3]);

    // allocation video memory for our dot product sum
    hipMalloc((void **)&d_c_dot, sizeof(int));
    hipMemcpy(d_c_dot, &cDot, sizeof(int), hipMemcpyHostToDevice);

    // vector dot product is done in 1 block as well
    vector_dot_product<<<1, N>>>(d_c, d_a, d_b);
    hipDeviceSynchronize();

    // copy back the final sum back to main memory
    hipMemcpy(&cDot, d_c, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_c);

    printf("[Dot Product]\tOutput of C: %d", cDot);

    return 0;
}